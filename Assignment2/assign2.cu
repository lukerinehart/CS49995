// Luke Rinehart
// GPU parallel processing Assignment 2
// GPU N average sort

#include <hip/hip_runtime.h>

#include <time.h>
#include <stdio.h>

const int size = 100;

void gpu_avg(int [size][size], int *,  int);

__global__ void n_avg(int A[size][size], int *B, int k){

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int sum = 0;
  int count = 0;
  printf("{%i}", row);

  if(row < size) {
   for(int i = 0; i < k; ++i) {
       sum += A[row][i];
       count = count + 1;
   }

   B[count] = sum;

 }

}

void gpu_avg(int A[size][size], int *B, int k){
   int Ad[size][size];
   int *Bd;

   dim3 blocks(32,32);
   dim3 grids(1,1);

   hipMalloc((void**)&Ad, k);
   hipMemcpy(Ad,A,k*k,hipMemcpyHostToDevice);
   hipMalloc((void**)&Bd, k);                        /*Allocate Space for A & B on Device*/
   hipMemcpy(Bd,B,k,hipMemcpyHostToDevice);

   n_avg<<<grids,blocks>>>(Ad,Bd,k);         /* Run Average*/

   hipFree(Ad);
   hipFree(Bd);  /* Free mem */

}

int main()
{
 //int size = 100;
 int A[size][size]; // 10,000 values
 int result[size];

 srand(time(0));
    for(int i = 0; i < size; ++i){
       result[i] = 0;
       for(int j = 0; j < size; ++j){
          A[i][j] =  rand() % 100;
        }
    }

 hipEvent_t start, stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);

 hipEventRecord(start);

 gpu_avg(A,result,size);

 hipEventRecord(stop);
 hipEventSynchronize(stop);

 float milliseconds = 0;
 hipEventElapsedTime(&milliseconds, start, stop);   /* Report Time */
 printf("%f ms\n", milliseconds);

 return 0;
}

